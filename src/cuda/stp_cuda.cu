#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include "eigen3/Eigen/Dense"
#include "eigen3/Eigen/Sparse"

#include <excute_cuda.hpp>


uint64_t Total_Thread = 0; // total supported threads


extern "C"
//get total thread number
void Get_Total_Thread_Num(void)
{
    int deviceCount;
    //get the number of CUDA devices
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess)
    {
        std::cerr << "hipGetDeviceCount failed: " << hipGetErrorString(error) << std::endl;
        return;
    }

    for (int i = 0; i < deviceCount; ++i) 
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "Max blocks per grid: " << (int64_t)deviceProp.maxGridSize[0] * deviceProp.maxGridSize[1] * deviceProp.maxGridSize[2] << std::endl;
        std::cout << "  X : " << deviceProp.maxGridSize[0] << std::endl;
        std::cout << "  Y : " << deviceProp.maxGridSize[1] << std::endl;
        std::cout << "  Z : " << deviceProp.maxGridSize[2] << std::endl;
        std::cout << "Total max threads: " << deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount << std::endl;
        std::cout << std::endl;
        Total_Thread = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount;
    }    
}




//In_KR_Matrix_Kernel     
__global__ void In_KR_Matrix_Kernel(int32_t sub_dim, int32_t idx_offset, int32_t *A, int32_t A_val_len, int32_t *C, int32_t C_val_len)
{
    int32_t ix = blockIdx.x * 1024 + threadIdx.x;
    //index
    int32_t idx = ix + idx_offset;

    int32_t x_code = idx / A_val_len;

    int32_t y_code = idx % A_val_len;

    //boundary check  
    if(idx < C_val_len)
    {    
        //XP+Y
        C[idx] = x_code * A[0] + A[y_code + 1]; 
    }
}


extern "C"
//In_KR_Matrix
std::vector<int32_t> cuda_In_KR_Matrix(int32_t dim, std::vector<int32_t>& A)
{
    //Get the dimensions of matrix A
    int32_t A_row = A[0];
    int32_t A_col = A.size() - 1;

    //Calculate the size of result matrix
    int32_t C_len = dim * A_col + 1;

    //Define the result matrix
    std::vector<int32_t> C(C_len);

    //Assign the number of rows of result matrix
    C[0] = A_row * dim;

    if (C_len <= Total_Thread / 5)
    {
        for (int32_t i = 0; i < dim; i++)
        {
            int32_t temp = i * A_row;
            int32_t idx = i * A_col + 1;
            for (int32_t j = 0; j < A_col; j++)
            {
                C[idx + j] = temp + A[j + 1];
            }
        }
        return C;
    }

    int32_t *d_A,*d_C;

    //compute space 
    size_t size_A = A.size() * sizeof(int32_t);
    size_t size_C = (C_len - 1) * sizeof(int32_t);

    //allocate memory
    hipError_t errA, errC;
    errA = hipMalloc((void **)&d_A, size_A);
    //Error checking
    if (errA != hipSuccess) 
    {
        std::cerr << "Error allocating memory for In_KR_Matrix d_A: " << hipGetErrorString(errA) << std::endl;

        std::vector<int32_t> Temp;
        //Set the size to improve the speed of computation
        Temp.resize(1);
        Temp[0] = -1;

        return Temp;
    }
    errC = hipMalloc((void **)&d_C, size_C);
    if (errC != hipSuccess) 
    {
        std::cerr << "Error allocating memory for In_KR_Matrix d_C: " << hipGetErrorString(errC) << std::endl;
        std::vector<int32_t> Temp;
        //Set the size to improve the speed of computation
        Temp.resize(1);
        Temp[0] = -1;

        return Temp;
    }

    //Copy parameters
    hipMemcpy(d_A, A.data(), size_A, hipMemcpyHostToDevice);

    //Calculate the block size (maximum 1024)
    dim3 threadsPerBlock(1024, 1);

    //Can be done in one go (with each element of C as a thread)
    if((C_len - 1) <=Total_Thread)
    {
        //Calculate the grid size (for large scale)
        dim3 numBlocks0(( C_len - 1 + 1024 -1 ) / 1024, 1);

        //Launch GPU
        In_KR_Matrix_Kernel<<<numBlocks0, threadsPerBlock>>>(dim, 0, d_A, A_col , d_C, C_len - 1);
        hipDeviceSynchronize(); //Wait for the kernel to complete
    }
    //Divide into blocks (by column)
    else
    {
        int32_t remain_num = C_len - 1; //remaining unassigned columns in C
        int32_t idx_offset = 0;  //Thread offset

        while(remain_num)
        {
            //the last  
            if(remain_num <= Total_Thread)
            {
                //Calculate the grid size (for large scale)
                dim3 numBlocks(( remain_num + 1024 -1 ) / 1024, 1);

                In_KR_Matrix_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, d_A, A_col, d_C, C_len - 1); 
                hipDeviceSynchronize(); //Wait for the kernel to complete
                //Calculate the thread offset
                idx_offset += remain_num;
                remain_num = 0; //exit the loop
            }
            // Total thread
            else
            {
                //Calculate the grid size (for large scale)
                dim3 numBlocks(( Total_Thread + 1024 -1 ) / 1024, 1);

                In_KR_Matrix_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, d_A, A_col, d_C, C_len - 1);  
                hipDeviceSynchronize(); //Wait for the kernel to complete

                //Calculate thread offset
                idx_offset += Total_Thread;
                remain_num -= Total_Thread; //exit the loop                
            }
        }     

    }

    //Segmented copy with pointer offset
    hipMemcpy(C.data() + 1, d_C, size_C, hipMemcpyDeviceToHost);

    //Free resources
    hipFree(d_A);
    hipFree(d_C);

    return C;
}



//Matrix_KR_In_Kernel
__global__ void Matrix_KR_In_Kernel(int32_t dim, int32_t idx_offset, int32_t *A, int32_t A_val_len, int32_t *C, int32_t C_val_len)
{
    int32_t ix = blockIdx.x * 1024 + threadIdx.x;
    //index
    int32_t idx = ix + idx_offset;

    //calculate x_code
    int32_t x_code = idx / dim;

    //calculate y_code
    int32_t y_code = idx % dim;

    //boundary check
    if(idx < C_val_len)
    {    
        //xp+y
        C[idx] = A[x_code] * dim + y_code;
    }
    
}

//Matrix_KR_In
std::vector<int32_t> cuda_Matrix_KR_In(int32_t dim,  std::vector<int32_t>& A)
{
    //get dimensions of matrix A
    int32_t A_row = A[0];
    int32_t A_col = A.size() - 1;

    //calculate size of result matrix
    //int32_t C_len = A_row * dim + 1;
    int32_t C_len = A_col * dim + 1;

    std::vector<int32_t> C(C_len);

    //assign number of rows of result matrix
    C[0] = A_row * dim;

    if (C_len <= Total_Thread / 5)
    {
        for (int32_t i = 0; i < A_col; i++)
        {
            int32_t temp = A[i + 1] * dim;
            int32_t idx = i * dim + 1;
            for (int32_t j = 0; j < dim; j++)
            {
                C[idx + j] = temp + j;
            }
        }
        return C;

    }

    int32_t *d_A,*d_C;
    //compute space
    size_t size_A = A.size() * sizeof(int32_t);
    size_t size_C = ( C_len - 1 ) * sizeof(int32_t);

    //allocate memory
    hipError_t errA, errC;

    errA = hipMalloc((void **)&d_A, size_A);
    //error checking
    if (errA != hipSuccess) 
    {
        std::cerr << "Error allocating memory for Matrix_KR_In d_A: " << hipGetErrorString(errA) << std::endl;
        std::vector<int32_t> Temp;
        //set size to improve computation speed
        Temp.resize(1);
        Temp[0] = -1;

        return Temp;
    }
    errC = hipMalloc((void **)&d_C, size_C);
    if (errC != hipSuccess) 
    {
        std::cerr << "Error allocating memory for Matrix_KR_In d_C: " << hipGetErrorString(errC) << std::endl;
        std::vector<int32_t> Temp;
        //set size to improve computation speed
        Temp.resize(1);
        Temp[0] = -1;

        return Temp;
    }

    //copy parameters
    hipMemcpy(d_A, A.data(), size_A, hipMemcpyHostToDevice);

    //calculate block size (maximum 1024)
    dim3 threadsPerBlock(1024, 1);

    //can be done in one go (with each element of C as a thread)
    if((C_len - 1) <= Total_Thread)
    {
        //calculate grid size (for large scale)
        dim3 numBlocks(( C_len - 1 + 1024 -1 ) / 1024, 1);

        Matrix_KR_In_Kernel<<<numBlocks, threadsPerBlock>>>(dim, 0, d_A + 1, A_col, d_C, C_len - 1); 
        hipDeviceSynchronize(); //wait for the kernel to complete
    }
    //divide into blocks (by column)
    else
    {
        int32_t remain_num = C_len - 1; //C remaining unassigned columns
        int32_t idx_offset = 0;  //thread offset

        while(remain_num)
        {
            //last time  quantity remain_num
            if(remain_num <= Total_Thread)
            {
                //calculate grid size (for large scale) 
                dim3 numBlocks(( remain_num + 1024 -1 ) / 1024, 1);

                Matrix_KR_In_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, d_A + 1, A_col, d_C, C_len - 1);  
                hipDeviceSynchronize(); //wait for the kernel to complete
                //calculate thread offset
                idx_offset += remain_num;
                remain_num = 0; //exit the loop
            }
            //total thread
            else
            {
                //calculate grid size (for large scale)
                dim3 numBlocks(( Total_Thread + 1024 -1 ) / 1024, 1);

                Matrix_KR_In_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, d_A + 1, A_col, d_C, C_len - 1);  
                hipDeviceSynchronize(); //wait for the kernel to complete

                //calculate thread offset
                idx_offset += Total_Thread;
                remain_num -= Total_Thread; //exit the loop                
            }
        }
    }

    //segmented copy with pointer offset
    hipMemcpy(C.data() + 1, d_C, size_C, hipMemcpyDeviceToHost);

    //free resources
    hipFree(d_A);
    hipFree(d_C);

    return C;
}         


//Matrix_Multipiy_Kernel
__global__ void Matrix_Multipiy_Kernel(int32_t idx_offset, int32_t *A, int32_t A_val_len, int32_t *B, int32_t B_val_len, int32_t *C, int32_t C_val_len, int32_t t)
{
    int32_t ix = blockIdx.x * 1024 + threadIdx.x;
    //index
    int32_t idx = ix + idx_offset;

    //calculate x_code
    int32_t x_code = idx % t ;

    //calculate y_code
    int32_t y_code = idx / t;

    //boundary check
    if(idx < C_val_len )
    {

        //result
        C[idx] = A[ B[ y_code + 1 ] * t + x_code + 1];
    }
}

extern "C"
//my_semi_tensor_product
std::vector<int32_t> cuda_semi_tensor_product(std::vector<int32_t>& A, std::vector<int32_t>& B)
{
    //get dimensions of matrix A and B
    int32_t A_row = A[0];
    int32_t A_col = A.size() - 1;
    int32_t B_row = B[0];
    int32_t B_col = B.size() - 1;

    if(A_col % B_row == 0)
    {
        //calculate size of result matrix
        int32_t C_len = (int64_t)A_col * B_col / B_row + 1;

        if (C_len <= Total_Thread / 5)
        {
            std::vector<int32_t> C(C_len);

            C[0] = A_row;
            int32_t times = A_col / B_row;

            for (int32_t i = 0; i < B_col; i++)
            {
                //t = n/p
                for(int32_t j = 0; j < times; j++)
                {
                    C[times * i + j + 1] = A[1 + B[i+1] * times + j] ;
                }
            }

            return C;
        }

        //caculate size of result matrix
        size_t size_A = A.size() * sizeof(int32_t);
        size_t size_B = B.size() * sizeof(int32_t);
        size_t size_C = (C_len - 1) * sizeof(int32_t);       

        int32_t *d_A,*d_B,*d_C;

        //allocate memory
        hipError_t errA, errB, errC;
        errA = hipMalloc((void **)&d_A, size_A);

        //error checking
        if (errA != hipSuccess) 
        {
            std::cerr << "Error allocating memory for my_semi_tensor_product d_A: " << hipGetErrorString(errA) << std::endl;
            std::vector<int32_t> Temp;
            //set size to improve computation speed
            Temp.resize(1);
            Temp[0] = -1;

            return Temp;
        }
        errB = hipMalloc((void **)&d_B, size_B);
        //error checking
        if (errB != hipSuccess) 
        {
            std::cerr << "Error allocating memory for my_semi_tensor_product d_B: " << hipGetErrorString(errB) << std::endl;
            std::vector<int32_t> Temp;
            //set size to improve computation speed
            Temp.resize(1);
            Temp[0] = -1;

            return Temp;
        }
        errC = hipMalloc((void **)&d_C, size_C);
        if (errC != hipSuccess) 
        {
            std::cerr << "Error allocating memory for my_semi_tensor_product d_C: " << hipGetErrorString(errC) << std::endl;
            std::vector<int32_t> Temp;
            //set size to improve computation speed
            Temp.resize(1);
            Temp[0] = -1;

            return Temp;
        }

        //copy parameters
        hipMemcpy(d_A, A.data(), size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B.data(), size_B, hipMemcpyHostToDevice);

        //calculate block size (maximum 1024)
        dim3 threadsPerBlock(1024, 1);

        if((C_len - 1) <= Total_Thread)
        {
            //calculate grid size (for large scale)
            dim3 numBlocks((C_len - 1 + 1024 - 2 ) / 1024, 1);
            Matrix_Multipiy_Kernel<<<numBlocks, threadsPerBlock>>>(0, d_A, A_col, d_B, B_col, d_C, C_len - 1, A_col / B_row);

            //wait for all threads to complete
            hipDeviceSynchronize();          
        }
        else
        {
            int32_t remain_num = C_len - 1; //remaining unassigned columns in C
            int32_t idx_offset = 0;  //thread offset

            while(remain_num)
            {
                //the last  
                if(remain_num <= Total_Thread)
                {
                    //calculate grid size (for large scale) 
                    dim3 numBlocks(( remain_num + 1024 -1 ) / 1024, 1);

                    Matrix_Multipiy_Kernel<<<numBlocks, threadsPerBlock>>>(idx_offset, d_A, A_col, d_B, B_col, d_C, C_len - 1, A_col / B_row);
                    hipDeviceSynchronize(); //wait for the kernel to complete
                    //calculate thread offset
                    idx_offset += remain_num;
                    remain_num = 0; //exit the loop
                }
                //total thread
                else
                {
                    //calculate grid size (for large scale)
                    dim3 numBlocks(( Total_Thread + 1024 -1 ) / 1024, 1);

                    Matrix_Multipiy_Kernel<<<numBlocks, threadsPerBlock>>>(idx_offset, d_A, A_col, d_B, B_col, d_C, C_len - 1, A_col / B_row);
                    hipDeviceSynchronize(); //wait for the kernel to complete
                    //calculate thread offset
                    //idx_offset += Total_Thread;
                    hipDeviceSynchronize(); //wait for the kernel to complete
                    //计算线程偏移
                    idx_offset += Total_Thread;
                    remain_num -= Total_Thread; //exit the loop               
                }
            }            
        }

        std::vector<int32_t> C;
        //set size to improve computation speed
        C.resize(C_len);

        //assign result matrix row number
        C[0] = A_row;

        //segmented copy with pointer offset
        hipMemcpy(C.data() + 1, d_C, size_C, hipMemcpyDeviceToHost);
        //free resources
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        return C;
    }
    else if(B_row % A_col == 0)
    {
        std::vector<int32_t> temp = cuda_Matrix_KR_In(B_row / A_col, A);
        std::vector<int32_t> C = cuda_semi_tensor_product(temp, B);

        return C;
    }
    else
    {
        //error
        std::cout << "Error" << std::endl;
        std::vector<int32_t> C;
        //set size to improve computation speed
        C.resize(1);
        C[0] = -1;

        return C;
    }
    
}




extern "C"
//my_chain_multiply_by_multi_thread                                                
std::vector<int32_t> cuda_chain_multiply_by_multi_thread( std::vector<std::vector<int32_t>>& mc, bool verbose)
{
  std::vector<int32_t> result=mc[0];

  if(mc.size()<2)
  {
    return result;
  }

  for (size_t i = 1; i < mc.size(); i++ )
  {
    result = cuda_semi_tensor_product(result,mc[i]);

  }
  return result;
}




